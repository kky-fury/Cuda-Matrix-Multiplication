#include "hip/hip_runtime.h"
#include "cuda_mmult_kernels.h"

/* 
 * matrix multiplication C += A*B 
 *  -> CUDA kernel
 *     (implementation adopted from Kirk&Hwu: 
 *      "Programming Massively Parallel Processors, chapter 4)
 *  -> Features: none (basic tiled version, using only global memory)
 */
__global__ void matrixMultKernel_global(float* Ad, float* Bd, float* Cd, int n)
{
   	int row = blockIdx.y*blockDim.y + threadIdx.y;
   	int column = blockIdx.x*blockDim.x + threadIdx.x;
   
   	if(row >=n || column >=n)
   	{
		return;
   	}
    
   	float Celem = 0.0;
   	for(int j = 0;j<n;j++)
   	{
        	Celem += *(Ad + row*n + j)*(*(Bd + j*n + column));
   	}
	
   	*(Cd + row*n + column) = Celem;

}

/* 
 * matrix multiplication C += A*B 
 *  -> CUDA kernel
 *     (implementation adopted from Kirk&Hwu: 
 *      "Programming Massively Parallel Processors, chapter 5)
 *  -> Features:
 *     - tiled matrix multiplication with use of shared memory
 */
__global__ void matrixMultKernel_tiled(float* Ad, float* Bd, float* Cd, int n)
{
	__shared__ float A_shared[TILE_SIZE][TILE_SIZE];
	__shared__ float B_shared[TILE_SIZE][TILE_SIZE];

	int row = blockIdx.y*TILE_SIZE + threadIdx.y;
	int column = blockIdx.x*TILE_SIZE + threadIdx.x;
	
	if(row >=n || column >=n)
	{	
		return;
	}
	
 	float Celem = 0.0;
	for(int m = 0; m < n/TILE_SIZE;m++)
	{
		*(*(A_shared + threadIdx.y) + threadIdx.x) = *(Ad + row*n + m*TILE_SIZE + threadIdx.x);
		*(*(B_shared + threadIdx.y) + threadIdx.x) = *(Bd + (m*TILE_SIZE + threadIdx.y)*n + column);
		__syncthreads();
		
		for(int k  = 0;k<TILE_SIZE;k++)
		{
			Celem += *(*(A_shared + threadIdx.y) + k)*(*(*(B_shared + k) + threadIdx.x));
		}
		__syncthreads();		

	} 
	*(Cd +  row*n + column) = Celem;
			
}

/* 
 * matrix multiplication C += A*B 
 *  -> CUDA kernel
 *     (implementation adopted from Kirk&Hwu: 
 *      "Programming Massively Parallel Processors, chapter 5)
 *  -> Features:
 *     - tiled matrix multiplication with use of shared memory
 *     - coalesced memory access
 *     - overlapping loads of subsequent tile pairs (using registers & shared memory)
 */

__global__ void matrixMultKernel_overlap(float* Ad, float* Bd, float* Cd, int n)

{
	__shared__ float A_shared[TILE_SIZE][TILE_SIZE];
	__shared__ float B_shared[TILE_SIZE][TILE_SIZE];
	
	int row = blockIdx.y*TILE_SIZE + threadIdx.y;
	int column = blockIdx.x*TILE_SIZE + threadIdx.x;

	if(row >= n || column >=n)
	{	
		return;
	}

	float Celem = 0.0;
	float reg_1 = *(Ad + row*n + threadIdx.x);
	float reg_2 = *(Bd + threadIdx.y*n + column);

	for(int m = 1;m<n/TILE_SIZE;m++)
	{
		A_shared[threadIdx.y][threadIdx.x] = reg_1;
		B_shared[threadIdx.y][threadIdx.x] = reg_2;

		__syncthreads();
		
		reg_1 = *(Ad + row*n + m*TILE_SIZE + threadIdx.x);
		reg_2 = *(Bd + (m*TILE_SIZE + threadIdx.y)*n + column);	
		
		for(int k = 0;k<TILE_SIZE;k++)
		{
			Celem += A_shared[threadIdx.y][k]*B_shared[k][threadIdx.x];
		}	 	

		__syncthreads();
	}
	
	A_shared[threadIdx.y][threadIdx.x] = reg_1;
	B_shared[threadIdx.y][threadIdx.x] = reg_2;
	
	__syncthreads();
	
	for(int k = 0;k<TILE_SIZE;k++)
	{
		Celem += A_shared[threadIdx.y][k]*B_shared[k][threadIdx.x];
	}
	
	__syncthreads();

	*(Cd + row*n + column) = Celem;
	
}
